
#include <hip/hip_runtime.h>
#include "stdio.h"

__global__ void add(int a, int b, int *c)
{
 c[threadIdx.x] = a + b;
}

int main()
{
    int a, b, c, blockSz;
    int *dev_c;
    a=3;
    b=4;
    blockSz = 10;
    hipMalloc((void**)&dev_c, sizeof(int) * blockSz);
    add<<<1, blockSz>>>(a, b, dev_c);
    hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost);
    printf("%d + %d is %d\n", a, b, c);
    hipFree(dev_c);
    return 0;
}